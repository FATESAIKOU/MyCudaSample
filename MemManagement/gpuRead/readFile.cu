#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "flex/fileParser.h"
}

__global__ void showRecord(char ***records, int rec_id, int attr_num) {
    int i;

    for (i = 0; i < attr_num; i ++) {
        printf("true!! : %s\n", records[rec_id][i]);
    }
}


__global__ void showMatchBase(MatchBase *match_base, int base) {
    int t_id = threadIdx.x;
    int b_id = blockIdx.x;
    int b_dim = blockDim.x;
    int index = base + b_id * b_dim + t_id;
    int j = 0;

    if (index < match_base->record_num) {
        printf("---record %d---\n", index);
        for (j = 0; j < 12; j ++) {
            printf("true!! : %s\n", match_base->sample_records[index][j]);
        }
        printf("\n");
    }
}

int recordCpy(char ***G_record, char **record, int attr_num) {
    int i;
    int attr_len;

    char **tmp_attrs;
    char *tmp_attr;
    hipMalloc(&tmp_attrs, sizeof(char*) * attr_num);
    for (i = 0; i < attr_num; i ++) {
        attr_len = strlen(record[i]);
        hipMalloc(&tmp_attr, sizeof(char) * attr_len);
        hipMemcpy(tmp_attr, record[i], sizeof(char) * attr_len, hipMemcpyHostToDevice);

        hipMemcpy(tmp_attrs + i, &tmp_attr, sizeof(char*), hipMemcpyHostToDevice);
    }

    hipMemcpy(G_record, &tmp_attrs, sizeof(char**), hipMemcpyHostToDevice);

    return attr_num;
}

int recordsCpy(char ****G_records, char ***records, int record_num) {
    int i;

    char ***tmp_records;
    hipMalloc(&tmp_records, sizeof(char**) * record_num);

    for (i = 0; i < record_num; i ++) {
        recordCpy(tmp_records + i, records[i], 11);
    }

    *G_records = tmp_records;
    return record_num;
}

int main(int argc, char *argv[]) {
    char *filename = argv[1];
    MatchBase match_base;

    readFile(filename, &match_base);

    // gen cloned G_match_base
    MatchBase *H_match_base = (MatchBase*) malloc(sizeof(MatchBase));
    H_match_base->record_num = match_base.record_num;
    recordsCpy(&(H_match_base->sample_records), match_base.sample_records, match_base.record_num);
    //H_match_base->record_num = 102400;
    //recordsCpy(&(H_match_base->sample_records), match_base.sample_records, 102400);

    // gen G_match_base
    MatchBase *G_match_base;
    hipMalloc(&G_match_base, sizeof(MatchBase));
    hipMemcpy(G_match_base, H_match_base, sizeof(MatchBase), hipMemcpyHostToDevice);

    // show data
    //showMatchBase<<<1, 1024>>>(G_match_base, 0);

    // reset
    //hipDeviceReset();

    // return & free
    hipFree(H_match_base);
    return 0;
}
