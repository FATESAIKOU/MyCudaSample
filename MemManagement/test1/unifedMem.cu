#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_LENGTH 1000000000

__global__ void print(char ***strs, int record_num, int col_num) {
    int t_id = threadIdx.x;
    int i;

    if (t_id < record_num) {
        printf("---t_id %d---\n", t_id);
        for (i = 0; i < col_num; i ++) {
            printf("\tattr %d: %s\n", i, strs[t_id][i]);
        }
    }
}
int main(int argc, char *argv[]) {
    char ***records;

    int i, j, record_num = 102400, col_num = 12;
    hipMallocManaged(&records, sizeof(char**) * record_num);

    int str_len;
    for (i = 0; i < record_num; i ++) {
        hipMallocManaged(&records[i], sizeof(char*) * col_num);

        for (j = 0; j < col_num; j ++) {
            str_len = strlen("Hello world\n");
            hipMallocManaged(&records[i][j], sizeof(char) * str_len);
            strcpy(records[i][j], "Hello world\n");
        }
    }

    hipDeviceSynchronize();

    print<<<1, 2>>>(records, record_num, col_num);

    hipDeviceReset();

    return 0;
}

