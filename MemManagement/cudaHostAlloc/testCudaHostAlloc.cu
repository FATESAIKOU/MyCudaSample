#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define ThreadNum 256

__global__ void printBase(int **base, int length) {
    int t_id = threadIdx.x;
    int b_id = blockIdx.x;

    if (t_id < length) {
        printf("block:%d-%d : %d\n", b_id, t_id, base[b_id][t_id]);
    }
}

int main(int agrc, char *argv[]) {
    int limit = atoi(argv[1]);
    int **base;

    hipMallocManaged(&base, sizeof(int*) * limit);
    hipDeviceSynchronize();

    int i, j;
    for (i = 0; i < limit; i ++) {
        hipMallocManaged(&base[i], sizeof(int) * 256);
        for (j = 0; j < ThreadNum; j ++) {
            base[i][j] = i * 1000 + j;
        }
    }

    int block_num = limit;
    printBase<<<block_num, ThreadNum>>>(base, ThreadNum);
    hipDeviceSynchronize();

    hipDeviceReset();
    hipFree(base);

    return 0;
}
