#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define THREADNUM 1024
#define BLOCKNUM 127

__device__ float G_rand(hiprandState *states, int ind){
    hiprandState local_state = states[ind];
    float rand_num = hiprand_uniform(&local_state);
    states[ind] = local_state;

    return rand_num;
}

__global__ void G_srand(hiprandState *states, unsigned long seed){
    int ind = threadIdx.x;
    //what is curand_init
    hiprand_init(seed, ind, 0, &states[ind]);
}

__global__ void getPoints(int *correct, int *base, int *block_result, int loop_time, hiprandState *states){
    int t_id = threadIdx.x;
    int b_id = blockIdx.x;
    int i;

    __shared__ int result[THREADNUM+1];

    if(t_id != 0){
        int current_count = 0;
        double pi_x;
        double pi_y;
        result[t_id] = 0;

        for(i=0; i<loop_time; i++){
            pi_x = G_rand(states, t_id);
            pi_y = G_rand(states, t_id);
            if( (pi_x*pi_x + pi_y*pi_y) < 1 )
                ++current_count;
        }
        result[t_id] = current_count;
    }

    __syncthreads();

    if(t_id == 0){
        int block_correct = 0;
        for(i=1; i<THREADNUM; i++)
            block_correct += result[i];

        block_result[b_id] = block_correct;
    }

    __syncthreads();

    if(!t_id && !b_id){
        *correct = 0;
        *base = loop_time*(THREADNUM-1)*BLOCKNUM;
        for(i=0; i<BLOCKNUM; i++)
            *correct += block_result[i];
    }
}

int main(){
    hiprandState *dev_states;
    hipMalloc((void**) &dev_states, sizeof(hiprandState) * THREADNUM);
    G_srand<<<1/*BLOCKNUM*/, THREADNUM>>>(dev_states, unsigned(time(NULL)));

    int *G_correct, *G_base, *G_block;;
    hipMalloc((void**) &G_correct, sizeof(int));
    hipMalloc((void**) &G_base, sizeof(int));
    hipMalloc((void**) &G_block, sizeof(int)*BLOCKNUM);

    int i;
    long double correct = 0, base = 0;
    for(i=0; i<10000; i++){
        getPoints<<<BLOCKNUM, THREADNUM>>>(G_correct, G_base, G_block, 10000, dev_states);

        int now_correct, now_base;
        hipMemcpy(&now_correct, G_correct, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&now_base, G_base, sizeof(int), hipMemcpyDeviceToHost);

        system("clear");
        printf("correct =\t%.0Lf\n", correct);
        printf("base =\t\t%.0Lf\n", base);

        correct += now_correct;
        base += now_base;
        printf("answer = %.15Lf\n", (correct/base) * 4);
    }

    hipFree(dev_states);
    hipFree(G_correct);
    hipFree(G_base);


    return 0;
}
