#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <unistd.h>
#include <string>

#include <hip/hip_runtime.h>

#define ThreadNum 256
#define BlockNum 16

__global__ void printOut(char *string) {
    printf("%s\n", string);
}

size_t getFileSize(char *filename) {
    struct stat st;
    stat(filename, &st);

    return st.st_size;
}

void parsing(char *aim, long int **offset_table, int *entry) {
    int limit = 1024;
    int i;

    long int *tmp_offset = (long int*) malloc(sizeof(long int) * limit);
    char *token = strtok(aim, "\n");
    for (i = 0; token != NULL; i ++) {
        if (i == limit) {
            limit += 1024;
            tmp_offset = (long int*) realloc(tmp_offset, sizeof(long int) * limit);
        }

        tmp_offset[i] = token - aim;
        token = strtok(NULL, "\n");
    }
    printf("Count %d\n", i);

    // realloc table
    tmp_offset = (long int*) realloc(tmp_offset, sizeof(long int) * i);

    // assign & return
    *offset_table = tmp_offset;
    *entry = i;
}

__device__ int strlen(char *s) {
    int i = 0;
    while (s[i] != '\0') i ++;

    return i;
}

__device__ char *strstrDevice(char *a, char *b) {
    int i, j;
    int a_len = strlen(a);
    int b_len = strlen(b);
    int loop_limit = a_len - b_len + 1;

    for (i = 0; i < loop_limit; i ++) {
        for (j = 0; j < b_len && a[i + j] == b[j]; j ++);

        if (j == b_len) return a + i;
    }

    return NULL;
}

__global__ void matching(char *aim, char *string, long int *offset_table, int entry, int base, int *result) {
    int t_id = threadIdx.x;
    int b_id = blockIdx.x;
    int b_dim = blockDim.x;

    int index = base + b_id * b_dim + t_id;
    //int aim_len = offset_table[index + 1] - offset_table[index];

    //if (index < entry && strstrDevice(string + offset_table[index], aim_len, "apple", 5) != NULL) {
    if (index < entry && strstrDevice(string + offset_table[index], aim) != NULL) {
        result[index] = 1;
    } else {
        result[index] = 0;
    }
}

int myCmp(const void *a, const void *b) {
    return (*(int*) a) - (*(int*) b);
}

int main(int argc, char *argv[]) {
    char *filename = argv[1];
    int fd = open(filename, O_RDONLY, 0644);

    // get mmap data
    size_t file_len = getFileSize(filename) + 1;
    char *filecontent = (char*) mmap(NULL,  file_len, PROT_READ | PROT_WRITE, MAP_PRIVATE, fd, 0);
    filecontent[file_len - 1] = '\0';

    // parsing
    long int *offset_table;
    int entry;
    parsing(filecontent, &offset_table, &entry);

    // copy data to device
    char *HD_filecontent;
    hipMalloc(&HD_filecontent, file_len);
    hipMemcpy(HD_filecontent, filecontent, file_len, hipMemcpyHostToDevice);

    // copy offset table to device
    long int *D_offset_table;
    hipMalloc(&D_offset_table, sizeof(long int) * entry);
    hipMemcpy(D_offset_table, offset_table, sizeof(long int) * entry, hipMemcpyHostToDevice);

    // matching
    int round_limit = ceil(entry / (float) (ThreadNum * BlockNum));
    int i;
    int *result;
    hipMallocManaged(&result, sizeof(int) * entry);
    char *aim;
    hipMallocManaged(&aim, sizeof(char) * 6);
    strcpy(aim, "apple");
    hipDeviceSynchronize();
    for (i = 0; i < round_limit; i ++) {
        matching<<<BlockNum, ThreadNum>>>(aim, HD_filecontent, D_offset_table, entry, i * ThreadNum * BlockNum, result);
    }
    hipDeviceSynchronize();

    qsort(result, entry, sizeof(int), myCmp);

    return 0;
}
