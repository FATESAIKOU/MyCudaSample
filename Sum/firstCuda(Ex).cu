#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATASIZE 1048756
#define THREADNUM 128

int data[DATASIZE];

void GenerateNumbers(int *numbers, int size){
    for(int i=0; i<size; i++){
        numbers[i] = 1;
    }
}

__global__ static void sumOfSquare(int *num, int *result){
    const int t_id = threadIdx.x;
    float tmp_size = DATASIZE/(float)THREADNUM;
    const int size = tmp_size==(int)tmp_size? (int)tmp_size:(int)tmp_size+1;

    int i, sum = 0;

    for(i=t_id*size; i<(t_id+1)*size; i++){
        sum += num[i]*num[i];
    }
    result[t_id] = sum;
}

int main(){
    GenerateNumbers(data, DATASIZE);

    int *gpudata, *result;
    clock_t start;

    hipMalloc((void**) &gpudata, sizeof(int)*DATASIZE);
    hipMalloc((void**) &result, sizeof(int)*THREADNUM);
    hipMemcpy(gpudata, data, sizeof(int)*DATASIZE, hipMemcpyHostToDevice);

    start = clock();

    sumOfSquare<<<1, THREADNUM, 0>>>(gpudata, result);

    int sum[THREADNUM];
    hipMemcpy(&sum, result, sizeof(int)*THREADNUM, hipMemcpyDeviceToHost);

    int final_sum = 0, i;
    for(i=0; i<THREADNUM; i++){
        final_sum += sum[i];
    }
    printf("GPU: the answer is %d(time: %ld)\n", final_sum, clock()-start);

    hipFree(gpudata);
    hipFree(result);

    start = clock();
    for(final_sum=0, i=0; i<DATASIZE; i++){
        final_sum += data[i]*data[i];
    }
    printf("CPU: the answer is %d(time: %ld)\n", final_sum, clock()-start);

    return 0;
}
