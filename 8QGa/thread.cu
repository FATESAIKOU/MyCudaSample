#include <stdio.h>
#include <stdlib.h>

// cuda include
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

typedef struct{
  int *gene;
  int fitness;
}Indiv;

__device__ float Grand(hiprandState *state){
  //get index
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  //gen local_state
  hiprandState local_state = state[index];

  //get rand_num
  float rand_num = hiprand_uniform(&local_state);

  //write back rand status
  state[index] = local_state;

  //return rand_num
  return rand_num;
}

__device__ int randLimit(int limit, hiprandState *state){
  float f_rand = Grand(state) * (limit + 1);
  return (int)f_rand;
}

__device__ void GSrand(hiprandState *state, unsigned int seed){
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  hiprand_init(seed, index, 0, &state[index]);
}

__device__ int getParent(Indiv *source_space, int CCE){
  int ans = 0;

  return ans;
}

__device__ Indiv newIndiv(Indiv *source_space, int CCE, int gene_size){
  Indiv new_indiv;

  // get parents
  int *father = source_space[getParent(source_space, CCE)].gene;
  int *mother = source_space[getParent(source_space, CCE)].gene;

  // get gene space
  new_indiv.gene = (int*)malloc(sizeof(int) * gene_size);

  // gen new gene

  return new_indiv;
}

__global__ void newGeneration(Indiv *pre_generation, Indiv *now_generation){
  ;
}
