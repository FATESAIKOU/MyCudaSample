#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATASIZE 1048756

int data[DATASIZE];

void GenerateNumbers(int *numbers, int size){
    for(int i=0; i<size; i++){
        numbers[i] = 1;
    }
}

__global__ static void sumOfSquare(int *num, int *result, int size){
    int i, sum = 0;

    for(i=0; i<size; i++){
        sum += num[i]*num[i];
    }

    *result = sum;
}

int main(int argc, char *argv[]){
    GenerateNumbers(data, DATASIZE);

    int *gpudata, *result;

    hipMalloc((void**) &gpudata, sizeof(int)*DATASIZE);
    hipMalloc((void**) &result, sizeof(int));
    hipMemcpy(gpudata, data, sizeof(int)*DATASIZE, hipMemcpyHostToDevice);

    sumOfSquare<<<1, atoi(argv[1]), 0>>>(gpudata, result, DATASIZE);

    int sum;
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);

    printf("the answer is %d\n", sum);
}

