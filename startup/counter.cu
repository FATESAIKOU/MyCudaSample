#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ static void getTime(clock_t *time){
    *time = clock();
}

int main(){
    clock_t *now_time, real_time;

    hipMalloc((void**) &now_time, sizeof(clock_t));
    getTime<<<1, 1, 0>>>(now_time);
    hipMemcpy(&real_time, now_time, sizeof(clock_t), hipMemcpyDeviceToHost);

    printf("clock(): %ld\n getTime(): %ld\n", clock(), real_time);

    return 0;
}
