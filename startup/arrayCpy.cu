#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void printMat(int *mat, int row, int column){
  int i, j;

  printf("\t");
  for(j=0; j<column; j++)
    printf("<%02d>\t", j);
  printf("\n");

  for(i=0; i<row; i++){
    printf("<%02d>\t", i);
    for(j=0; j<column; j++)
      printf("%2d\t", mat[i*column+j]);

    printf("\n");
  }
}

int main(){
  int *a, *b, *c, *ac;
  int i, j;
  size_t ac_pitch;
  a = (int*)malloc(sizeof(int)*10*10);
  b = (int*)malloc(sizeof(int)*10*10);
  c = (int*)malloc(sizeof(int)*10*10);

  for(i=0; i<10; i++)
    for(j=0; j<10; j++){
      a[i*10+j] = i*10+j;
      b[i*10+j] = -1;
    }

  hipMallocPitch((void**) &ac, &ac_pitch, sizeof(int)*10, 10);
  printf("ac_pitch: %d\nsizeof(int)*10: %ld\n", ac_pitch, sizeof(int)*10);

  printf("\nOri data>>\n");
  printMat((int*)a, 10, 10);
  //partial copy to device
  hipMemcpy2D(ac, ac_pitch, a, sizeof(int)*10, sizeof(int)*10, 10, hipMemcpyHostToDevice);

  //fully copy to host
  hipMemcpy2D(b, sizeof(int)*10, ac, ac_pitch, sizeof(int)*10, 10, hipMemcpyDeviceToHost);
  printf("\nDevice data>>\n");
  printMat((int*)b, 10, 10);

  //partial copy to host
  hipMemcpy2D(c, sizeof(int)*10, ac, 36, sizeof(int)*10, 10, hipMemcpyDeviceToHost);
  printf("\npartial Copy>>\n");
  printMat((int*)c, 10, 10);

  return 0;
}
