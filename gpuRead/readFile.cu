extern "C" {
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "flex/fileParser.h"

int recordCpy(char ***G_records, char **record, int attr_num) {
    int i;
    int attr_len;

    char **tmp_attrs;
    char *tmp_attr;
    hipMalloc(&tmp_attrs, sizeof(char*) * attr_num);
    for (i = 0; i < attr_num; i ++) {
        attr_len = strlen(record[i]);
        hipMalloc(&tmp_attr, sizeof(char) * attr_len);
        hipMemcpy(tmp_attr, record[i], sizeof(char) * attr_len, hipMemcpyHostToDevice);

        hipMemcpy(tmp_attrs + i, &tmp_attr, sizeof(char*), hipMemcpyHostToDevice);
    }

    hipMemcpy(G_records, &tmp_attrs, sizeof(char**), hipMemcpyHostToDevice);

    return attr_num;
}
}

int main(int argc, char *argv[]) {
    char *filename = argv[1];
    MatchBase match_base;

    readFile(filename, &match_base);

    int i;
    char ***G_records;
    hipMalloc(&G_records, sizeof(char**) * match_base.record_num);
    for (i = 0; i < match_base.record_num; i ++) {
        recordCpy(G_records + i, match_base.sample_records[i], COLUMN_NUM);
    }


    MatchBase *G_match_base, tmp_G_match_base;
    hipMalloc(&G_match_base, sizeof(MatchBase));
    tmp_G_match_base.sample_records = G_records;
    tmp_G_match_base.record_num = match_base.record_num;

    hipMemcpy(G_match_base, &tmp_G_match_base, sizeof(MatchBase), hipMemcpyHostToDevice);
    return 0;
}
