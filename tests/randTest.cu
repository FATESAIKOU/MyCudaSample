#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define THREADNUM 4
#define BLOCKNUM 4

__device__ float G_rand(hiprandState *states, int ind){
    hiprandState local_state = states[ind];
    float rand_num = hiprand_uniform(&local_state);
    //states[ind] = local_state;

    return rand_num;
}

__global__ void G_srand(hiprandState *states, unsigned long seed){
    int ind = threadIdx.x;
    //what is curand_init
    hiprand_init(seed, ind, 0, &states[ind]);
}

__global__ void G_testRand(double *tmp_space, hiprandState *states){
  int t_id = threadIdx.x;
  int b_id = blockIdx.x;

  tmp_space[(b_id * THREADNUM) + t_id] = G_rand(states, t_id);

  return;
}

int main(){

    // initialize for parallel computation
    hiprandState *dev_states;
    hipMalloc((void**) &dev_states, sizeof(hiprandState) * THREADNUM);
    G_srand<<<BLOCKNUM, THREADNUM>>>(dev_states, unsigned(time(NULL)));

    // prepering for args space
    double *G_rand, *C_rand;
    hipMalloc((void**) &G_rand, sizeof(double) * BLOCKNUM * THREADNUM);
    C_rand = (double*)malloc(sizeof(double) * BLOCKNUM * THREADNUM);

    // calculation
    G_testRand<<<BLOCKNUM, THREADNUM>>>(G_rand, dev_states);

    // copy back to MainMemory
    hipMemcpy(C_rand, G_rand, sizeof(double) * BLOCKNUM * THREADNUM, hipMemcpyDeviceToHost);

    // output result
    int i, j;
    printf("Result: ----------------\n");
    for (i = 0; i < BLOCKNUM; i++) {
      for (j = 0; j < THREADNUM; j++) {
        printf("%lf\t", C_rand[(i * THREADNUM) + j]);
      }
      printf("\n");
    }

    // delete used memory
    hipFree(dev_states);
    hipFree(G_rand);
    free(C_rand);

    return 0;
}
