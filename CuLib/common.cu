#include <stdio.h>
#include <stdlib.h>

//cuda include
#include <hip/hip_runtime.h>

__device__ void Gswap(void *from, void *to, int length){
  void *tmp = malloc(length);

  memcpy(tmp, to, length);
  memcpy(to, from, length);
  memcpy(from, tmp, length);
}
