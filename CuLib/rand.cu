#include <stdio.h>
#include <stdlib.h>

// cuda include
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ float Grand(hiprandState *state){
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  hiprandState local_state = state[index];
  float rand_num = hiprand_uniform(&local_state);
  state[index] = local_state;

  return rand_num;
}

__device__ int GrandInt(hiprandState *state, int limit){
  float rand_num = Grand(state) * (limit + 1);

  return (int)rand_num;
}

__global__ void GSrand(hiprandState *state, unsigned int seed){
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  hiprand_init(seed, index, 0, &state[index]);
}
