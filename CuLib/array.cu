#include <stdio.h>
#include <stdlib.h>

//cuda include
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "common.h"
#include "rand.h"

__device__ void *arrayShuffle(void *array, int ele_size, int length, hiprandState *state){
  int i, rand_num;

  for(i=0; i<length; ++i){
    rand_num = GrandInt(state, length);
    Gswap((char*) array + i, (char*) array + rand_num, ele_size);
  }

  return array;
}
