#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void){
    hipDeviceProp_t prop;

    if(hipGetDeviceProperties(&prop, 0) != hipSuccess){
        fprintf(stderr, "no device avalible, go a hell, man\n");
    }

    fprintf(stdout, "get prop success\n");
    fprintf(stdout, "   name: %s\n", prop.name);
    fprintf(stdout, "   totalGlobalMem: %lu\n", prop.totalGlobalMem);
    fprintf(stdout, "   sharedMemPerBlock: %lu\n", prop.sharedMemPerBlock);
    fprintf(stdout, "   regsPerBlock: %d\n", prop.regsPerBlock);
    fprintf(stdout, "   warpSize: %d\n", prop.warpSize);
    fprintf(stdout, "   memPitch: %lu\n", prop.memPitch);
    fprintf(stdout, "   maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
    fprintf(stdout, "   maxThreadsDim:\n");
    fprintf(stdout, "       maxThreadsDim[0]: %d\n", prop.maxThreadsDim[0]);
    fprintf(stdout, "       maxThreadsDim[1]: %d\n", prop.maxThreadsDim[1]);
    fprintf(stdout, "       maxThreadsDim[2]: %d\n", prop.maxThreadsDim[2]);
    fprintf(stdout, "   maxThreadsGridSize:\n");
    fprintf(stdout, "       maxGridSize[0]: %d\n", prop.maxGridSize[0]);
    fprintf(stdout, "       maxGridSize[1]: %d\n", prop.maxGridSize[1]);
    fprintf(stdout, "       maxGridSize[2]: %d\n", prop.maxGridSize[2]);
    fprintf(stdout, "   clockRate: %d\n", prop.clockRate);
    fprintf(stdout, "   totalConstMem: %lu\n", prop.totalConstMem);
    fprintf(stdout, "   major.minor: %d.%d\n", prop.major, prop.minor);
    fprintf(stdout, "   textureAlignment: %lu\n", prop.textureAlignment);
    fprintf(stdout, "   texturePitchAlignment: %lu\n", prop.texturePitchAlignment);
    fprintf(stdout, "   deviceOverlap: %d\n", prop.deviceOverlap);
    fprintf(stdout, "   multiProcessorCount: %d\n", prop.multiProcessorCount);
    fprintf(stdout, "   kernelExecTimeoutEnabled: %d\n", prop.kernelExecTimeoutEnabled);
    fprintf(stdout, "   integrated: %d\n", prop.integrated);
    fprintf(stdout, "   canMapHostMemory: %d\n", prop.canMapHostMemory);
    fprintf(stdout, "   computeMode: %d\n", prop.computeMode);
    fprintf(stdout, "   maxTexture1D: %d\n", prop.maxTexture1D);
    fprintf(stdout, "   maxTexture1DLinear: %d\n", prop.maxTexture1DLinear);
    fprintf(stdout, "   maxTexture2D\n");
    fprintf(stdout, "       maxTexture2D[0]: %d\n", prop.maxTexture2D[0]);
    fprintf(stdout, "       maxTexture2D[1]: %d\n", prop.maxTexture2D[1]);
    fprintf(stdout, "   maxTexture2DLinear\n");
    fprintf(stdout, "       maxTexture2DLinear[0]: %d\n", prop.maxTexture2DLinear[0]);
    fprintf(stdout, "       maxTexture2DLinear[1]: %d\n", prop.maxTexture2DLinear[1]);
    fprintf(stdout, "       maxTexture2DLinear[2]: %d\n", prop.maxTexture2DLinear[2]);
    fprintf(stdout, "   maxTexture2DGather\n");
    fprintf(stdout, "       maxTexture2DGather[0]: %d\n", prop.maxTexture2DGather[0]);
    fprintf(stdout, "       maxTexture2DGather[1]: %d\n", prop.maxTexture2DGather[1]);
    fprintf(stdout, "   maxTexture3D\n");
    fprintf(stdout, "       maxTexture3D[0]: %d\n", prop.maxTexture3D[0]);
    fprintf(stdout, "       maxTexture3D[1]: %d\n", prop.maxTexture3D[1]);
    fprintf(stdout, "       maxTexture3D[2]: %d\n", prop.maxTexture3D[2]);
    fprintf(stdout, "   maxSurface1D: %d\n", prop.maxSurface1D);
    fprintf(stdout, "   maxSurface2D\n");
    fprintf(stdout, "       maxSurface2D[0]: %d\n",  prop.maxSurface2D[0]);
    fprintf(stdout, "       maxSurface2D[1]: %d\n",  prop.maxSurface2D[1]);
    fprintf(stdout, "   maxSurface3D\n");
    fprintf(stdout, "       maxSueface3D[0]: %d\n", prop.maxSurface3D[0]);
    fprintf(stdout, "       maxSueface3D[1]: %d\n", prop.maxSurface3D[1]);
    fprintf(stdout, "       maxSueface3D[2]: %d\n", prop.maxSurface3D[2]);
    fprintf(stdout, "   maxSurface1DLayered\n");
    fprintf(stdout, "       maxSurface1DLayered[0]: %d\n", prop.maxSurface1DLayered[0]);
    fprintf(stdout, "       maxSurface1DLayered[1]: %d\n", prop.maxSurface1DLayered[1]);
    fprintf(stdout, "   maxSurface2DLayered\n");
    fprintf(stdout, "       maxSurface2DLayered[0]: %d\n", prop.maxSurface2DLayered[0]);
    fprintf(stdout, "       maxSurface2DLayered[1]: %d\n", prop.maxSurface2DLayered[1]);
    fprintf(stdout, "       maxSurface2DLayered[2]: %d\n", prop.maxSurface2DLayered[2]);
    fprintf(stdout, "   maxSurfaceCubemap: %d\n", prop.maxSurfaceCubemap);
    fprintf(stdout, "   maxSurfaceCubemapLayered\n");
    fprintf(stdout, "       maxSurfaceCubemapLayered[0]: %d\n", prop.maxSurfaceCubemapLayered[0]);
    fprintf(stdout, "       maxSurfaceCubemapLayered[1]: %d\n", prop.maxSurfaceCubemapLayered[1]);
    fprintf(stdout, "   surfaceAlignment: %ld\n", prop.surfaceAlignment);
    fprintf(stdout, "   concurrentKernels: %d\n", prop.concurrentKernels);
    fprintf(stdout, "   ECCEnabled: %d\n", prop.ECCEnabled);
    fprintf(stdout, "   pciBusID: %d\n", prop.pciBusID);
    fprintf(stdout, "   pciDeviceID: %d\n", prop.pciDeviceID);
    fprintf(stdout, "   pciDomainID: %d\n", prop.pciDomainID);
    fprintf(stdout, "   tccDriver: %d\n", prop.tccDriver);
    fprintf(stdout, "   asyncEngineCount: %d\n", prop.asyncEngineCount);
    fprintf(stdout, "   unifiedAddressing: %d\n", prop.unifiedAddressing);
    fprintf(stdout, "   memoryClockRate: %d\n", prop.memoryClockRate);
    fprintf(stdout, "   l2CacheSize: %d\n", prop.l2CacheSize);
    fprintf(stdout, "   maxThreadsPerMultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
    return 0;
}
