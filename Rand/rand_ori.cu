#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define THREADNUM 512

__device__ float randGpu(hiprandState *global_state, int ind){
  //int ind = threadIdx.x;
  hiprandState local_state = global_state[ind];
  float rand_num = hiprand_uniform(&local_state);
  global_state[ind] = local_state;

  return rand_num;
}

__global__ void setupKernel(hiprandState *states, unsigned long seed){
  int ind = threadIdx.x;
  hiprand_init(seed, ind, 0, &states[ind]);
}

__global__ void genRandom(float *data, hiprandState *global_state){
  int ind = threadIdx.x;
  data[ind] = randGpu(global_state, ind);
}

int main(){
  float *data, *G_data;
  data = (float*)malloc(sizeof(float) * THREADNUM);
  hipMalloc((void**) &G_data, sizeof(float) * THREADNUM);

  hiprandState *dev_states;
  hipMalloc((void**) &dev_states, sizeof(hiprandState) * THREADNUM);

  setupKernel<<<1, THREADNUM>>>(dev_states, unsigned(time(NULL)));
  genRandom<<<1, THREADNUM>>>(G_data, dev_states);

  hipMemcpy(data, G_data, sizeof(float) * THREADNUM, hipMemcpyDeviceToHost);

  int i;
  for(i=0; i<THREADNUM; i++){
    printf("%f\n", data[i]);
  }

  return 0;
}
