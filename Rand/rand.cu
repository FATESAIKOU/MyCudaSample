#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//cuda include
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
/*
#define BLOCKNUM 100
#define THREADNUM 150
*/
__global__ void GSrand(hiprandState *state, unsigned int seed){
  int index = blockIdx.x * blockDim.x * threadIdx.x;

  hiprand_init(seed, index, 0, &state[index]);
}

__device__ float Grand(hiprandState *state){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState local_state = state[index];

  float rand_num = hiprand_uniform(&local_state);

  state[index] = local_state;

  return rand_num;
}

__global__ void testRand(float *rand_data, hiprandState *state){
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  rand_data[index] = Grand(state);
}

int main(int argc, char *argv[]){
  int BLOCKNUM = atoi(argv[1]);
  int THREADNUM = atoi(argv[2]);

  //data initialization
  int data_length = BLOCKNUM * THREADNUM;
  hiprandState *dev_state;
  hipMalloc((void**) &dev_state, sizeof(hiprandState) * data_length);

  //random initialization
  GSrand<<<BLOCKNUM, THREADNUM>>>(dev_state, (unsigned int)time(NULL));

  //malloc host & device data
  float *host_rand_data = (float*)malloc(sizeof(float) * data_length);
  float *dev_rand_data;
  hipMalloc((void**) &dev_rand_data, sizeof(float) * data_length);

  //get rand data
  testRand<<<BLOCKNUM, THREADNUM>>>(dev_rand_data, dev_state);

  //cpy data from dev to host
  hipMemcpy((void*) host_rand_data, (const void*) dev_rand_data, sizeof(float) * data_length, hipMemcpyDeviceToHost);

  //output result
  printf("RAND RESULT:~~\n");
  int i;
  for(i=0; i<data_length; ++i){
    printf("%d: %f\n", i, host_rand_data[i]);
  }

  return 0;
}
