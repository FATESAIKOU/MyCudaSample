#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MATSIZE 10000
#define THREADNUM 256

void matgen(int *mat, int size){
  srand((unsigned)time(NULL));

  int i, j;
  for(i=0; i<size; i++){
    for(j=0; j<size; j++)
      mat[i*size+j] = rand()%10;
  }
}

void printMat(int *mat, int size){
  int i, j;

  for(i=0; i<size; i++){
    printf("%02d>>\t", i);
    for(j=0; j<size; j++)
      printf("%4d\t", mat[i*MATSIZE+j]);
    printf("\n");
  }
  printf("\n");
}

clock_t matmultCPU(int *mat1, int *mat2, int *matR, int size){
  int i, j, k, tmp;
  clock_t start = clock();

  for(i=0; i<MATSIZE; i++)
    for(j=0; j<MATSIZE; j++){
      tmp = 0;
      for(k=0; k<MATSIZE; k++)
        tmp += mat1[i*size+k]*mat2[k*size+j];
      matR[i*size+j] = tmp;
  }

  clock_t end = clock();
  return end-start;
}

 __global__ static void multiGPU(int *mat1, size_t ld1, int *mat2, size_t ld2, int *matR, size_t ldR, int size){
  const int tid     = threadIdx.x;
  const int bid     = blockIdx.x;
  const int idx     = bid*blockDim.x+tid;
  const int row     = idx / size;
  const int column  = idx % size;

  int i = 0;
  if(row < size && column < size){
    int tmp_product = 0;

    for(i=0; i<size; i++){
      tmp_product += mat1[row*ld1 + i]*mat2[i*ld2 + column];
    }

    matR[row*ldR + column] = tmp_product;
  }
}

clock_t matmultGPU(int *mat1, size_t ld1, int *mat2, size_t ld2, int *matR, size_t ldR, int size){
  int *G_mat1, *G_mat2, *G_matR;
  clock_t start, end;

  start = clock();
  hipMalloc((void**) &G_mat1, sizeof(int)*size*size);
  hipMalloc((void**) &G_mat2, sizeof(int)*size*size);
  hipMalloc((void**) &G_matR, sizeof(int)*size*size);

  hipMemcpy2D(G_mat1, sizeof(int)*size, mat1, sizeof(int)*size, sizeof(int)*size, size, hipMemcpyHostToDevice);
  hipMemcpy2D(G_mat2, sizeof(int)*size, mat2, sizeof(int)*size, sizeof(int)*size, size, hipMemcpyHostToDevice);

  int blocks = (size+THREADNUM-1)/THREADNUM;
  multiGPU<<<blocks*size, THREADNUM>>>(G_mat1, ld1, G_mat2, ld2, G_matR, ldR, size);

  hipMemcpy2D(matR, sizeof(int)*size, G_matR, sizeof(int)*size, sizeof(int)*size, size, hipMemcpyDeviceToHost);

  hipFree(G_mat1);
  hipFree(G_mat2);
  hipFree(G_matR);
  end = clock();

  return end-start;
}

int main(int argc, char *argv[]){
  int *mat1, *mat2, *matR, *matRGPU;
  clock_t CPUused, GPUused;

  mat1 = (int*)malloc(sizeof(int)*MATSIZE*MATSIZE);
  mat2 = (int*)malloc(sizeof(int)*MATSIZE*MATSIZE);
  matR = (int*)malloc(sizeof(int)*MATSIZE*MATSIZE);
  matRGPU = (int*)malloc(sizeof(int)*MATSIZE*MATSIZE);

  matgen(mat1, MATSIZE);
  matgen(mat2, MATSIZE);

  if(argc > 1 && argv[1][0] == 'G'){
    GPUused = matmultGPU(mat1, sizeof(int)*MATSIZE, mat2, sizeof(int)*MATSIZE, matRGPU, sizeof(int)*MATSIZE, MATSIZE);
    printf("GPU time used: %ld\n", GPUused);
  }else{
    CPUused = matmultCPU(mat1, mat2, matR, MATSIZE);
    printf("CPU time used: %ld\n", CPUused);
  }

  return 0;
}
